#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "solution.h"



__global__ void kernel(float* a, float* b, float* c, int rows, int cols) {
	int tid  = threadIdx.x + blockIdx.x * blockDim.x;
	int i=0;
	float temp = 0;
	for(i =0; i<cols; i++){
		temp += a[i]*b[i*rows+tid];
	}
	c[tid] = temp;
	
}